#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>

#include "utils.cuh"
#include "gfx/display.cuh"
#include "gfx/renderer.cuh"
#include "gfx/camera.cuh"
#include "math/vec3.cuh"
#include "math/ray.cuh"
#include "objects/render_list.cuh"
#include "objects/sphere.cuh"

using namespace gfx;
using namespace math;
using namespace obj;
typedef unsigned int uint;

__device__ Vec3 calculate_color(const Ray& ray, RenderObject* world)
{
    HitResult result;
    if(world->hit(ray, 0.0f, FLT_MAX, result))
    {
        return 0.5f * (result.normal + Vec3::one());
    }
    else
    {
        const Vec3& direction = ray.direction();
        float t = 0.5f * (direction.y() + 1.0f);
        return (1.0f - t) * Vec3::one() + t * Vec3(0.5f, 0.7f, 1.0f);
    }
}

__global__ void create_world(RenderObject** world)
{
    if(threadIdx.x == 0 && blockIdx.x == 0)
    {
        RenderObject** objects = new RenderObject*[3];
        objects[0] = new Sphere(Vec3(1.0f, 0.0f, -1.0f), 0.5f);
        objects[1] = new Sphere(Vec3(-1.0f, 0.0f, -1.0f), 0.5f);
        objects[2] = new Sphere(Vec3(0, -100.5f, -1.0f), 100.0f);

        *world = new RenderList(objects, 3);
    }
}

__global__ void render(RenderObject** world, Camera camera, int width, int height, float4* pixels)
{
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= width || y >= height) return;

    float u = float(x) / float(width - 1);
    float v = float(y) / float(height - 1);
    Ray ray = camera.calculate_ray(u, v);

    Vec3 col = calculate_color(ray, *world);

    pixels[x + y * width].x = col.r();
    pixels[x + y * width].y = col.g();
    pixels[x + y * width].z = col.b();
}

dim3 calculate_grid_size(int width, int height, dim3 block_size)
{
    return dim3(
        width / block_size.x + (width % block_size.x == 0 ? 0 : 1),
        height / block_size.y + (height % block_size.y == 0 ? 0 : 1)
    );
}

int main(int argc, char* argv[])
{
    int window_width = 1400;
    int window_height = 700;
    int resolution_width = window_width;
    int resolution_height = window_height;
    std::string window_title = "CUDA Raytracer - Konrad Brzozka - Procesory Graficzne w Zastosowaniach Obliczeniowych";

    Display display(window_title, window_width, window_height);
    Renderer renderer(resolution_width, resolution_height);

    dim3 block_size(32, 32);
    dim3 grid_size = calculate_grid_size(resolution_width, resolution_height, block_size);

    RenderObject** world;
    CUDA_CHECK(hipMalloc(&world, sizeof(RenderObject*)));
    create_world<<<1, 1>>>(world);
    CUDA_CHECK(hipDeviceSynchronize());

    Camera camera(3, Vec3(0, 0, -1), 90, (float)resolution_width / resolution_height);
    float mouse_x = 0, mouse_y = 0;
    while(!display.should_close())
    {
        float new_mouse_x, new_mouse_y;
        display.get_cursor_pos(new_mouse_x, new_mouse_y);

        float dx = new_mouse_x - mouse_x;
        float dy = new_mouse_y - mouse_y;
        mouse_x = new_mouse_x;
        mouse_y = new_mouse_y;

        if(display.is_dragging())
            camera.move(-dx / 100, dy / 100);

        renderer.clear();

        float4* framebuffer = renderer.get_framebuffer();
        render<<<grid_size, block_size>>>(world, camera, resolution_width, resolution_height, framebuffer);
        CUDA_CHECK(hipDeviceSynchronize());

        renderer.blit();
        renderer.draw();
        display.show();
    }

    return 0;
}
