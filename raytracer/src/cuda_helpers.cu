#include "cuda_helpers.cuh"

void cuda_check(hipError_t result, const char* func, const char* file, const int line)
{
    if(result == hipSuccess) return;

    hipDeviceReset();

    std::stringstream ss;
    ss << "CUDA error " << (unsigned int)result << " at " << file << ':' << line
        << " in '" << func << "': " << hipGetErrorString(result);
    throw std::runtime_error(ss.str());
}
